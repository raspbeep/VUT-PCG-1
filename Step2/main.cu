#include "hip/hip_runtime.h"
/**
 * @file      main.cu
 *
 * @author    Name Surname \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xlogin00@fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2024
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <cmath>
#include <cstdio>
#include <chrono>
#include <string>

#include "nbody.cuh"
#include "h5Helper.h"

/**
 * @brief CUDA error checking macro
 * @param call CUDA API call
 */
#define CUDA_CALL(call) \
  do { \
    const hipError_t _error = (call); \
    if (_error != hipSuccess) \
    { \
      std::fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString(_error)); \
      std::exit(EXIT_FAILURE); \
    } \
  } while(0)

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  if (argc != 10)
  {
    std::printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    std::exit(1);
  }

  // Number of particles
  const unsigned N                   = static_cast<unsigned>(std::stoul(argv[1]));
  // Length of time step
  const float    dt                   = std::stof(argv[2]);
  // Number of steps
  const unsigned steps               = static_cast<unsigned>(std::stoul(argv[3]));
  // Number of thread blocks
  const unsigned simBlockDim         = static_cast<unsigned>(std::stoul(argv[4]));
  // Write frequency
  const unsigned writeFreq           = static_cast<unsigned>(std::stoul(argv[5]));
  // number of reduction threads
  const unsigned redTotalThreadCount = static_cast<unsigned>(std::stoul(argv[6]));
  // Number of reduction threads/blocks
  const unsigned redBlockDim         = static_cast<unsigned>(std::stoul(argv[7]));

  // Size of the simulation CUDA grid - number of blocks
  const unsigned simGridDim = (N + simBlockDim - 1) / simBlockDim;
  // Size of the reduction CUDA grid - number of blocks
  const unsigned redGridDim = (redTotalThreadCount + redBlockDim - 1) / redBlockDim;

  // Log benchmark setup
  std::printf("       NBODY GPU simulation\n"
              "N:                       %u\n"
              "dt:                      %f\n"
              "steps:                   %u\n"
              "threads/block:           %u\n"
              "blocks/grid:             %u\n"
              "reduction threads/block: %u\n"
              "reduction blocks/grid:   %u\n",
              N, dt, steps, simBlockDim, simGridDim, redBlockDim, redGridDim);

  const std::size_t recordsCount = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  
  const size_t allocSize = N * sizeof(float);
  
  Particles hParticles{};

  /********************************************************************************************************************/
  /*                              TODO: CPU side memory allocation (pinned)                                           */
  /********************************************************************************************************************/
  CUDA_CALL(hipHostAlloc(&hParticles.position_x, allocSize, hipHostMallocDefault));
  CUDA_CALL(hipHostAlloc(&hParticles.position_y, allocSize, hipHostMallocDefault));
  CUDA_CALL(hipHostAlloc(&hParticles.position_z, allocSize, hipHostMallocDefault));

  CUDA_CALL(hipHostAlloc(&hParticles.velocity_x, allocSize, hipHostMallocDefault));
  CUDA_CALL(hipHostAlloc(&hParticles.velocity_y, allocSize, hipHostMallocDefault));
  CUDA_CALL(hipHostAlloc(&hParticles.velocity_z, allocSize, hipHostMallocDefault));

  CUDA_CALL(hipHostAlloc(&hParticles.mass, allocSize, hipHostMallocDefault));

  /********************************************************************************************************************/
  /*                              TODO: Fill memory descriptor layout                                                 */
  /********************************************************************************************************************/
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                            Stride of two            Offset of the first
   *       Data pointer       consecutive elements        element in FLOATS,
   *                          in FLOATS, not bytes            not bytes
  */
  MemDesc md(hParticles.position_x,     1,                        0,
             hParticles.position_y,     1,                        0,
             hParticles.position_z,     1,                        0,
             hParticles.velocity_x,     1,                        0,
             hParticles.velocity_y,     1,                        0,
             hParticles.velocity_z,     1,                        0,
             hParticles.mass,           1,                        0,
             N,
             recordsCount);

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::fprintf(stderr, "Error: %s\n", e.what());
    return EXIT_FAILURE;
  }

  Particles dParticles[2]{};

  /********************************************************************************************************************/
  /*                                     TODO: GPU side memory allocation                                             */
  /********************************************************************************************************************/
  CUDA_CALL(hipMalloc(&dParticles[0].position_x, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[0].position_y, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[0].position_z, allocSize));
  
  CUDA_CALL(hipMalloc(&dParticles[0].velocity_x, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[0].velocity_y, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[0].velocity_z, allocSize));
  
  CUDA_CALL(hipMalloc(&dParticles[0].mass, allocSize));

  CUDA_CALL(hipMalloc(&dParticles[1].position_x, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[1].position_y, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[1].position_z, allocSize));

  CUDA_CALL(hipMalloc(&dParticles[1].velocity_x, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[1].velocity_y, allocSize));
  CUDA_CALL(hipMalloc(&dParticles[1].velocity_z, allocSize));

  CUDA_CALL(hipMalloc(&dParticles[1].mass, allocSize));

  /********************************************************************************************************************/
  /*                                     TODO: Memory transfer CPU -> GPU                                             */
  /********************************************************************************************************************/
  CUDA_CALL(hipMemcpy(dParticles[0].position_x, hParticles.position_x, allocSize, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles[0].position_y, hParticles.position_y, allocSize, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles[0].position_z, hParticles.position_z, allocSize, hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(dParticles[0].velocity_x, hParticles.velocity_x, allocSize, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles[0].velocity_y, hParticles.velocity_y, allocSize, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles[0].velocity_z, hParticles.velocity_z, allocSize, hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(dParticles[0].mass, hParticles.mass, allocSize, hipMemcpyHostToDevice));

  // can be copied in device
  CUDA_CALL(hipMemcpy(dParticles[1].position_x, dParticles[0].position_x, allocSize, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpy(dParticles[1].position_y, dParticles[0].position_y, allocSize, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpy(dParticles[1].position_z, dParticles[0].position_z, allocSize, hipMemcpyDeviceToDevice));

  CUDA_CALL(hipMemcpy(dParticles[1].velocity_x, dParticles[0].velocity_x, allocSize, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpy(dParticles[1].velocity_y, dParticles[0].velocity_y, allocSize, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpy(dParticles[1].velocity_z, dParticles[0].velocity_z, allocSize, hipMemcpyDeviceToDevice));

  CUDA_CALL(hipMemcpy(dParticles[1].mass, dParticles[0].mass, allocSize, hipMemcpyDeviceToDevice));

  /********************************************************************************************************************/
  /*                                  TODO: Set dynamic shared memory computation                                     */
  /********************************************************************************************************************/
  const std::size_t sharedMemSize = simBlockDim * sizeof(float) * 7;

  // Start measurement
  const auto start = std::chrono::steady_clock::now();

  for (unsigned s = 0u; s < steps; ++s)
  {
    const unsigned srcIdx = s % 2;        // source particles index
    const unsigned dstIdx = (s + 1) % 2;  // destination particles index

    /******************************************************************************************************************/
    /*                   TODO: GPU kernel invocation with correctly set dynamic memory size                           */
    /******************************************************************************************************************/
    calculateVelocity<<<simGridDim, simBlockDim, sharedMemSize>>>(dParticles[srcIdx], dParticles[dstIdx], N, dt);
  }

  // Wait for all CUDA kernels to finish
  CUDA_CALL(hipDeviceSynchronize());

  // End measurement
  const auto end = std::chrono::steady_clock::now();

  // Approximate simulation wall time
  const float elapsedTime = std::chrono::duration<float>(end - start).count();
  std::printf("Time: %f s\n", elapsedTime);

  const unsigned resIdx = steps % 2;    // result particles index

  /********************************************************************************************************************/
  /*                                     TODO: Memory transfer GPU -> CPU                                             */
  /********************************************************************************************************************/
  CUDA_CALL(hipMemcpy(hParticles.position_x, dParticles[resIdx].position_x, allocSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.position_y, dParticles[resIdx].position_y, allocSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.position_z, dParticles[resIdx].position_z, allocSize, hipMemcpyDeviceToHost));
  
  CUDA_CALL(hipMemcpy(hParticles.velocity_x, dParticles[resIdx].velocity_x, allocSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velocity_y, dParticles[resIdx].velocity_y, allocSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velocity_z, dParticles[resIdx].velocity_z, allocSize, hipMemcpyDeviceToHost));

  CUDA_CALL(hipMemcpy(hParticles.mass, dParticles[resIdx].mass, allocSize, hipMemcpyDeviceToHost));

  // Compute reference center of mass on CPU
  const float4 refCenterOfMass = centerOfMassRef(md);

  std::printf("Reference center of mass: %f, %f, %f, %f\n",
              refCenterOfMass.x,
              refCenterOfMass.y,
              refCenterOfMass.z,
              refCenterOfMass.w);

  std::printf("Center of mass on GPU: %f, %f, %f, %f\n", 0.f, 0.f, 0.f, 0.f);

  // Writing final values to the file
  h5Helper.writeComFinal(refCenterOfMass);
  h5Helper.writeParticleDataFinal();

  /********************************************************************************************************************/
  /*                                     TODO: GPU side memory deallocation                                           */
  /********************************************************************************************************************/
  CUDA_CALL(hipFree(dParticles[0].position_x));
  CUDA_CALL(hipFree(dParticles[0].position_y));
  CUDA_CALL(hipFree(dParticles[0].position_z));

  CUDA_CALL(hipFree(dParticles[0].velocity_x));
  CUDA_CALL(hipFree(dParticles[0].velocity_y));
  CUDA_CALL(hipFree(dParticles[0].velocity_z));

  CUDA_CALL(hipFree(dParticles[0].mass));

  CUDA_CALL(hipFree(dParticles[1].position_x));
  CUDA_CALL(hipFree(dParticles[1].position_y));
  CUDA_CALL(hipFree(dParticles[1].position_z));

  CUDA_CALL(hipFree(dParticles[1].velocity_x));
  CUDA_CALL(hipFree(dParticles[1].velocity_y));
  CUDA_CALL(hipFree(dParticles[1].velocity_z));
  
  CUDA_CALL(hipFree(dParticles[1].mass)); 
  
  /********************************************************************************************************************/
  /*                                     TODO: CPU side memory deallocation                                           */
  /********************************************************************************************************************/
  CUDA_CALL(hipHostFree(hParticles.position_x));
  CUDA_CALL(hipHostFree(hParticles.position_y));
  CUDA_CALL(hipHostFree(hParticles.position_z));

  CUDA_CALL(hipHostFree(hParticles.velocity_x));
  CUDA_CALL(hipHostFree(hParticles.velocity_y));
  CUDA_CALL(hipHostFree(hParticles.velocity_z));

  CUDA_CALL(hipHostFree(hParticles.mass));

}// end of main
//----------------------------------------------------------------------------------------------------------------------
