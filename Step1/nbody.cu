#include "hip/hip_runtime.h"
/**
 * @file      nbody.cu
 *
 * @author    Name Surname \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xlogin00@fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2024
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <>
#include <cfloat>
#include "nbody.cuh"

/* Constants */
constexpr float G                  = 6.67384e-11f;
constexpr float COLLISION_DISTANCE = 0.01f;

/**
 * CUDA kernel to calculate new particles velocity and position
 * @param pIn  - particles in
 * @param pOut - particles out
 * @param N    - Number of particles
 * @param dt   - Size of the time step
 */
__global__ void calculateVelocity(Particles pIn, Particles pOut, const unsigned N, float dt)
{
  /********************************************************************************************************************/
  /*          TODO: CUDA kernel to calculate new particles velocity and position, collapse previous kernels           */
  /********************************************************************************************************************/
  const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N)
    return;

  float newVelX{};
  float newVelY{};
  float newVelZ{};

  const float posX   = pIn.position_x[idx];
  const float posY   = pIn.position_y[idx];
  const float posZ   = pIn.position_z[idx];
  
  const float velX   = pIn.velocity_x[idx];
  const float velY   = pIn.velocity_y[idx];
  const float velZ   = pIn.velocity_z[idx];

  const float weight = pIn.mass[idx];
  
  for (unsigned i = 0; i < N; i++)
  {
    if (i == idx)
      continue;

    const float otherPosX = pIn.position_x[i];
    const float otherPosY = pIn.position_y[i];
    const float otherPosZ = pIn.position_z[i];
    
    const float otherVelX = pIn.velocity_x[i];
    const float otherVelY = pIn.velocity_y[i];
    const float otherVelZ = pIn.velocity_z[i];
    
    const float otherWeight = pIn.mass[i];

    const float dx = otherPosX - posX;
    const float dy = otherPosY - posY;
    const float dz = otherPosZ - posZ;

    const float r2 = dx * dx + dy * dy + dz * dz;
    const float r = sqrtf(r2);

    // Calculate gravitation velocity
    const float F = G * weight * otherWeight / (r2 + FLT_MIN);
    newVelX += (r > COLLISION_DISTANCE) ? dx / r * F : 0.f;
    newVelY += (r > COLLISION_DISTANCE) ? dy / r * F : 0.f;
    newVelZ += (r > COLLISION_DISTANCE) ? dz / r * F : 0.f;

    // Calculate collision velocity
    if (r > 0.f && r < COLLISION_DISTANCE)
    {
      newVelX += (((weight * velX - otherWeight * velX + 2.f * otherWeight * otherVelX) / (weight + otherWeight)) - velX);
      newVelY += (((weight * velY - otherWeight * velY + 2.f * otherWeight * otherVelY) / (weight + otherWeight)) - velY);
      newVelZ += (((weight * velZ - otherWeight * velZ + 2.f * otherWeight * otherVelZ) / (weight + otherWeight)) - velZ);
    }
  }

  newVelX *= dt / weight;
  newVelY *= dt / weight;
  newVelZ *= dt / weight;

  pOut.velocity_x[idx] = newVelX;
  pOut.velocity_y[idx] = newVelY;
  pOut.velocity_z[idx] = newVelZ;

  // Update particle positions and velocities
  float posXUpdated = posX + (velX + newVelX) * dt;
  float posYUpdated = posY + (velY + newVelY) * dt;
  float posZUpdated = posZ + (velZ + newVelZ) * dt;

  pOut.position_x[idx] = posXUpdated;
  pOut.position_y[idx] = posYUpdated;
  pOut.position_z[idx] = posZUpdated;

  pOut.velocity_x[idx] += newVelX;
  pOut.velocity_y[idx] += newVelY;
  pOut.velocity_z[idx] += newVelZ;
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate particles center of mass
 * @param p    - particles
 * @param com  - pointer to a center of mass
 * @param lock - pointer to a user-implemented lock
 * @param N    - Number of particles
 */
__global__ void centerOfMass(Particles p, float4* com, int* lock, const unsigned N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassRef(MemDesc& memDesc)
{
  float4 com{};

  for (std::size_t i{}; i < memDesc.getDataSize(); i++)
  {
    const float3 pos = {memDesc.getPosX(i), memDesc.getPosY(i), memDesc.getPosZ(i)};
    const float  w   = memDesc.getWeight(i);

    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    // Calculate weight ratio only if at least one particle isn't massless
    const float4 d = {pos.x - com.x,
                      pos.y - com.y,
                      pos.z - com.z,
                      ((memDesc.getWeight(i) + com.w) > 0.0f)
                        ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w))
                        : 0.0f};

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += d.x * d.w;
    com.y += d.y * d.w;
    com.z += d.z * d.w;
    com.w += w;
  }

  return com;
}// enf of centerOfMassRef
//----------------------------------------------------------------------------------------------------------------------
